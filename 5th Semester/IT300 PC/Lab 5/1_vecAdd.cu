#define N 256


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h> 

__global__ void vecAdd (int *a, int *b, int *c);
void printArray(int a[N], int b[N], int c[N]);

int main() {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	clock_t t;
	double time_taken;

	FILE *fp;
    fp = fopen ("output.txt","a");

	// initialize a and b with real values (NOT SHOWN)
	int size = N * sizeof(int);
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i/2;
	}

	t = clock();

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	hipMemcpy(dev_a, a, size,hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size,hipMemcpyHostToDevice);

	vecAdd<<<1,N>>>(dev_a,dev_b,dev_c);
	
	hipMemcpy(c, dev_c, size,hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	t = clock() - t;
	time_taken = ((double)t)/CLOCKS_PER_SEC;
	printf("fun() took %lf seconds to execute \n", time_taken); 

	fprintf (fp, "%d %lf\n", N, time_taken);

	printArray(a,b,c);

	fclose(fp);
	
	exit (0);
}

__global__ void vecAdd (int *a, int *b, int *c) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

void printArray(int a[], int b[], int c[]) {

	printf("Array a:\n");
	for(int i = 0; i < N; i++){
		printf("%d ", a[i]);
    }
    printf("\n\nArray b:\n");
    for(int i = 0; i < N; i++){
		printf("%d ", b[i]);
    }
    printf("\n\nArray c:\n");
    for(int i = 0; i < N; i++){
		printf("%d ", c[i]);
	}
	printf("\n");
}
