#include "hip/hip_runtime.h"
#define N 1024
#define T 256 // max threads per block

#include <stdio.h>

__global__ void vecAdd (int *a, int *b, int *c);
void printArray(int a[], int b[], int c[]);

int main() {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	clock_t t;
	double time_taken;

	FILE *fp;
    fp = fopen ("output.txt","a");

	// initialize a and b with real values (NOT SHOWN)
	int size = N * sizeof(int);
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i/2;
	}

	t = clock();

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	hipMemcpy(dev_a, a, size,hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size,hipMemcpyHostToDevice);

	vecAdd<<<(int)ceil(N/T),T>>>(dev_a,dev_b,dev_c);

	hipMemcpy(c, dev_c, size,hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	t = clock() - t;
	time_taken = ((double)t)/CLOCKS_PER_SEC;
	printf("Vector addition with 256 threads per block and padding for array of length %d took %lf seconds to execute \n", N, time_taken); 

	fprintf (fp, "%d %lf\n", N, time_taken);

	// printArray(a,b,c);

	fclose(fp);

	exit (0);
}

__global__ void vecAdd (int *a, int *b, int *c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		c[i] = a[i] + b[i];
	}
}

void printArray(int a[], int b[], int c[]) {

	printf("Array a:\n");
	for(int i = 0; i < N; i++){
		printf("%d ", a[i]);
    }
    printf("\n\nArray b:\n");
    for(int i = 0; i < N; i++){
		printf("%d ", b[i]);
    }
    printf("\n\nArray c:\n");
    for(int i = 0; i < N; i++){
		printf("%d ", c[i]);
	}
	printf("\n");
}
