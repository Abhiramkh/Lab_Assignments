#include "hip/hip_runtime.h"
#include<stdio.h>
#define N 1280
#define BLOCK_DIM 16

__global__ void matrixAdd (int *a, int *b, int *c);
void printArray(int a[N][N], int b[N][N], int c[N][N]);

int main() {
    int a[N][N], b[N][N], c[N][N];
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            a[i][j] = i+j;
            b[i][j] = i-j;
            c[i][j] = 0;
        }
    }
    int *dev_a, *dev_b, *dev_c;
    int size = N * N * sizeof(int);
    clock_t t;
	double time_taken;

    FILE *fp;
    fp = fopen ("output.txt","a");

    t = clock();

    // initialize a and b with real values (NOT SHOWN)
    hipMalloc((void**) &dev_a, size);
    hipMalloc((void**) &dev_b, size);
    hipMalloc((void**) &dev_c, size);

    hipMemcpy (dev_a, a, size, hipMemcpyHostToDevice) ;
    hipMemcpy (dev_b, b, size, hipMemcpyHostToDevice) ;
    
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM) ;
    dim3 dimGrid( (int) ceil (N/dimBlock.x) , (int) ceil (N/dimBlock.y));
    
    matrixAdd<<<dimGrid, dimBlock>>> (dev_a,dev_b,dev_c);
    
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
    hipFree(dev_a); 
    hipFree(dev_b); 
    hipFree(dev_c);

    t = clock() - t;
	time_taken = ((double)t)/CLOCKS_PER_SEC;
	printf("fun() took %lf seconds to execute \n", time_taken); 

	fprintf (fp, "%d %lf\n", N, time_taken);

    // printArray(a,b,c);

    fclose(fp);

    exit (0);
}

__global__ void matrixAdd (int *a, int *b, int *e) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col + row * N;
    if (col < N && row < N) {
        e[index] = a[index] + b[index] ;
        //printf("a[i][j] = %d, b[i][j] = %d, i,j = %d,%d\n", a[i][j], b[i][j], i, j);
    }
}

void printArray(int a[N][N], int b[N][N], int c[N][N]) {

    printf("Array a:\n");
	for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", a[i][j]);
        }
        printf("\n");
    }
    printf("\n\nArray b:\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", b[i][j]);
        }
        printf("\n");
    }
    printf("\n\nArray c:\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }
    printf("\n");

}
